#include "hip/hip_runtime.h"
#include "gpu_engine.hpp"
#include <stdexcept>
#include <cstring>

namespace graph_engine {

// CUDA kernel declarations
namespace kernels {

template<typename T>
__global__ void vertexKernel(vertex_id_t* row_offsets,
                            vertex_id_t* column_indices,
                            weight_t* weights,
                            T* vertex_data,
                            size_t num_vertices);

template<typename T>
__global__ void edgeKernel(vertex_id_t* column_indices,
                          weight_t* weights,
                          T* edge_data,
                          size_t num_edges);

} // namespace kernels

// Constants
constexpr int THREADS_PER_BLOCK = 256;
constexpr int MIN_GPU_COMPUTE_CAPABILITY = 35; // Minimum required: 3.5

GPUEngine::GPUEngine() 
    : device_id_(0)
    , initialized_(false)
{
    initialize();
}

GPUEngine::~GPUEngine() {
    if (gpu_data_) {
        if (gpu_data_->row_offsets) hipFree(gpu_data_->row_offsets);
        if (gpu_data_->column_indices) hipFree(gpu_data_->column_indices);
        if (gpu_data_->weights) hipFree(gpu_data_->weights);
    }
}

void GPUEngine::initialize() {
    if (initialized_) return;

    // Check for CUDA device
    int device_count;
    checkCudaError(hipGetDeviceCount(&device_count),
                   "Failed to get CUDA device count");

    if (device_count == 0) {
        throw std::runtime_error("No CUDA-capable devices found");
    }

    // Set device
    checkCudaError(hipSetDevice(device_id_),
                   "Failed to set CUDA device");

    if (!checkGPUCapabilities()) {
        throw std::runtime_error("GPU capabilities insufficient");
    }

    gpu_data_ = std::make_unique<GPUGraphData>();
    initialized_ = true;
}

void GPUEngine::transferGraphToDevice(const Graph* graph) {
    if (!graph) {
        throw std::invalid_argument("Null graph pointer");
    }

    // Convert graph to CSR format
    std::vector<vertex_id_t> row_offsets;
    std::vector<vertex_id_t> column_indices;
    std::vector<weight_t> weights;

    row_offsets.reserve(graph->getVertexCount() + 1);
    column_indices.reserve(graph->getEdgeCount());
    if (graph->isWeighted()) {
        weights.reserve(graph->getEdgeCount());
    }

    size_t current_offset = 0;
    for (vertex_id_t i = 0; i < graph->getVertexCount(); ++i) {
        row_offsets.push_back(current_offset);
        const Vertex* vertex = graph->getVertex(i);
        if (vertex) {
            for (const auto& [target, edge_id] : vertex->getOutEdges()) {
                column_indices.push_back(target);
                if (graph->isWeighted()) {
                    const Edge* edge = graph->getEdge(edge_id);
                    weights.push_back(edge ? edge->getWeight() : 1.0f);
                }
                current_offset++;
            }
        }
    }
    row_offsets.push_back(current_offset);

    // Allocate GPU memory
    size_t vertex_size = (graph->getVertexCount() + 1) * sizeof(vertex_id_t);
    size_t edge_size = graph->getEdgeCount() * sizeof(vertex_id_t);
    size_t weight_size = graph->isWeighted() ? graph->getEdgeCount() * sizeof(weight_t) : 0;

    checkCudaError(hipMalloc(&gpu_data_->row_offsets, vertex_size),
                   "Failed to allocate GPU memory for row offsets");
    checkCudaError(hipMalloc(&gpu_data_->column_indices, edge_size),
                   "Failed to allocate GPU memory for column indices");
    
    if (graph->isWeighted()) {
        checkCudaError(hipMalloc(&gpu_data_->weights, weight_size),
                      "Failed to allocate GPU memory for weights");
    }

    // Copy data to GPU
    checkCudaError(hipMemcpy(gpu_data_->row_offsets, row_offsets.data(),
                             vertex_size, hipMemcpyHostToDevice),
                   "Failed to copy row offsets to GPU");
    checkCudaError(hipMemcpy(gpu_data_->column_indices, column_indices.data(),
                             edge_size, hipMemcpyHostToDevice),
                   "Failed to copy column indices to GPU");
    
    if (graph->isWeighted()) {
        checkCudaError(hipMemcpy(gpu_data_->weights, weights.data(),
                                 weight_size, hipMemcpyHostToDevice),
                      "Failed to copy weights to GPU");
    }

    // Store graph properties
    gpu_data_->num_vertices = graph->getVertexCount();
    gpu_data_->num_edges = graph->getEdgeCount();
    gpu_data_->is_directed = graph->isDirected();
    gpu_data_->is_weighted = graph->isWeighted();
}

template<typename T>
void GPUEngine::parallelVertexOp(std::function<void(vertex_id_t, T&)> kernel,
                                std::vector<T>& vertex_data) {
    if (!initialized_ || !gpu_data_) {
        throw std::runtime_error("GPU engine not properly initialized");
    }

    // Allocate device memory for vertex data
    T* d_vertex_data;
    size_t vertex_data_size = vertex_data.size() * sizeof(T);
    checkCudaError(hipMalloc(&d_vertex_data, vertex_data_size),
                   "Failed to allocate GPU memory for vertex data");

    // Copy input data to device
    checkCudaError(hipMemcpy(d_vertex_data, vertex_data.data(),
                             vertex_data_size, hipMemcpyHostToDevice),
                   "Failed to copy vertex data to GPU");

    // Calculate grid dimensions
    int num_blocks = (gpu_data_->num_vertices + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    kernels::vertexKernel<<<num_blocks, THREADS_PER_BLOCK>>>(
        gpu_data_->row_offsets,
        gpu_data_->column_indices,
        gpu_data_->weights,
        d_vertex_data,
        gpu_data_->num_vertices
    );

    // Check for kernel launch errors
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    // Copy results back to host
    checkCudaError(hipMemcpy(vertex_data.data(), d_vertex_data,
                             vertex_data_size, hipMemcpyDeviceToHost),
                   "Failed to copy results from GPU");

    // Cleanup
    hipFree(d_vertex_data);
}

template<typename T>
void GPUEngine::parallelEdgeOp(std::function<void(edge_id_t, T&)> kernel,
                              std::vector<T>& edge_data) {
    if (!initialized_ || !gpu_data_) {
        throw std::runtime_error("GPU engine not properly initialized");
    }

    // Allocate device memory for edge data
    T* d_edge_data;
    size_t edge_data_size = edge_data.size() * sizeof(T);
    checkCudaError(hipMalloc(&d_edge_data, edge_data_size),
                   "Failed to allocate GPU memory for edge data");

    // Copy input data to device
    checkCudaError(hipMemcpy(d_edge_data, edge_data.data(),
                             edge_data_size, hipMemcpyHostToDevice),
                   "Failed to copy edge data to GPU");

    // Calculate grid dimensions
    int num_blocks = (gpu_data_->num_edges + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    kernels::edgeKernel<<<num_blocks, THREADS_PER_BLOCK>>>(
        gpu_data_->column_indices,
        gpu_data_->weights,
        d_edge_data,
        gpu_data_->num_edges
    );

    // Check for kernel launch errors
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    // Copy results back to host
    checkCudaError(hipMemcpy(edge_data.data(), d_edge_data,
                             edge_data_size, hipMemcpyDeviceToHost),
                   "Failed to copy results from GPU");

    // Cleanup
    hipFree(d_edge_data);
}

template<typename T>
T* GPUEngine::allocateDevice(size_t count) {
    T* device_ptr;
    checkCudaError(hipMalloc(&device_ptr, count * sizeof(T)),
                   "Failed to allocate GPU memory");
    return device_ptr;
}

template<typename T>
void GPUEngine::freeDevice(T* device_ptr) {
    if (device_ptr) {
        hipFree(device_ptr);
    }
}

template<typename T>
void GPUEngine::copyToDevice(const std::vector<T>& host_data, T* device_data) {
    checkCudaError(hipMemcpy(device_data, host_data.data(),
                             host_data.size() * sizeof(T), hipMemcpyHostToDevice),
                   "Failed to copy data to GPU");
}

template<typename T>
void GPUEngine::copyToHost(const T* device_data, std::vector<T>& host_data) {
    checkCudaError(hipMemcpy(host_data.data(), device_data,
                             host_data.size() * sizeof(T), hipMemcpyDeviceToHost),
                   "Failed to copy data from GPU");
}

bool GPUEngine::isGPUAvailable() const {
    int device_count;
    hipGetDeviceCount(&device_count);
    return device_count > 0;
}

int GPUEngine::getComputeCapability() const {
    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, device_id_),
                   "Failed to get device properties");
    return prop.major * 10 + prop.minor;
}

size_t GPUEngine::getAvailableMemory() const {
    size_t free_memory, total_memory;
    checkCudaError(hipMemGetInfo(&free_memory, &total_memory),
                   "Failed to get memory info");
    return free_memory;
}

void GPUEngine::checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        throw std::runtime_error(std::string(message) + ": " + 
                               hipGetErrorString(error));
    }
}

bool GPUEngine::checkGPUCapabilities() {
    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, device_id_),
                   "Failed to get device properties");
    
    int compute_capability = prop.major * 10 + prop.minor;
    return compute_capability >= MIN_GPU_COMPUTE_CAPABILITY;
}

// CUDA Kernel Implementations
namespace kernels {

template<typename T>
__global__ void vertexKernel(vertex_id_t* row_offsets,
                            vertex_id_t* column_indices,
                            weight_t* weights,
                            T* vertex_data,
                            size_t num_vertices) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_vertices) return;

    // Process vertex with ID 'tid'
    int start = row_offsets[tid];
    int end = row_offsets[tid + 1];
    
    for (int i = start; i < end; i++) {
        vertex_id_t neighbor = column_indices[i];
        weight_t weight = weights ? weights[i] : 1.0f;
        // Custom vertex processing logic here
    }
}

template<typename T>
__global__ void edgeKernel(vertex_id_t* column_indices,
                          weight_t* weights,
                          T* edge_data,
                          size_t num_edges) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_edges) return;

    // Process edge with ID 'tid'
    vertex_id_t target = column_indices[tid];
    weight_t weight = weights ? weights[tid] : 1.0f;
    // Custom edge processing logic here
}

} // namespace kernels

} // namespace graph_engine