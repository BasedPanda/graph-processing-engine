#include "hip/hip_runtime.h"
#include "gpu_engine.hpp"
#include <stdexcept>
#include <cstring>
#include "../utils/logger.hpp"

namespace graph_engine {

// CUDA kernel declarations
namespace kernels {

template<typename T>
__global__ void vertexKernel(
    vertex_id_t* row_offsets,
    vertex_id_t* column_indices,
    weight_t* weights,
    T* vertex_data,
    size_t num_vertices)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_vertices) return;

    // Process vertex with ID 'tid'
    const int start = row_offsets[tid];
    const int end = row_offsets[tid + 1];
    
    for (int i = start; i < end; i++) {
        vertex_id_t neighbor = column_indices[i];
        weight_t weight = weights ? weights[i] : 1.0f;
        // Custom vertex processing logic here
    }
}

template<typename T>
__global__ void edgeKernel(
    vertex_id_t* column_indices,
    weight_t* weights,
    T* edge_data,
    size_t num_edges)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_edges) return;

    // Process edge with ID 'tid'
    vertex_id_t target = column_indices[tid];
    weight_t weight = weights ? weights[tid] : 1.0f;
    // Custom edge processing logic here
}

} // namespace kernels

// Constants
constexpr int THREADS_PER_BLOCK = 256;
constexpr int MIN_GPU_COMPUTE_CAPABILITY = 35; // Minimum required: 3.5

GPUEngine::GPUEngine() 
    : device_id_(0)
    , initialized_(false)
{
    initialize();
}

GPUEngine::~GPUEngine() {
    if (gpu_data_) {
        if (gpu_data_->row_offsets) {
            hipFree(gpu_data_->row_offsets);
        }
        if (gpu_data_->column_indices) {
            hipFree(gpu_data_->column_indices);
        }
        if (gpu_data_->weights) {
            hipFree(gpu_data_->weights);
        }
    }
}

void GPUEngine::initialize() {
    if (initialized_) return;

    // Check for CUDA device
    int device_count;
    checkCudaError(hipGetDeviceCount(&device_count),
                   "Failed to get CUDA device count");

    if (device_count == 0) {
        throw std::runtime_error("No CUDA-capable devices found");
    }

    // Set device
    checkCudaError(hipSetDevice(device_id_),
                   "Failed to set CUDA device");

    if (!checkGPUCapabilities()) {
        throw std::runtime_error("GPU capabilities insufficient");
    }

    gpu_data_ = std::make_unique<GPUGraphData>();
    initialized_ = true;

    // Log GPU information
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id_);
    LOG_INFO("Using GPU: {} (Compute Capability {}.{})",
             prop.name, prop.major, prop.minor);
}

void GPUEngine::transferGraphToDevice(const Graph* graph) {
    if (!graph) {
        throw std::invalid_argument("Null graph pointer");
    }

    // Convert graph to CSR format
    std::vector<vertex_id_t> row_offsets;
    std::vector<vertex_id_t> column_indices;
    std::vector<weight_t> weights;

    row_offsets.reserve(graph->getVertexCount() + 1);
    column_indices.reserve(graph->getEdgeCount());
    if (graph->isWeighted()) {
        weights.reserve(graph->getEdgeCount());
    }

    size_t current_offset = 0;
    for (vertex_id_t i = 0; i < graph->getVertexCount(); ++i) {
        row_offsets.push_back(current_offset);
        const Vertex* vertex = graph->getVertex(i);
        if (vertex) {
            for (const auto& [target, edge_id] : vertex->getOutEdges()) {
                column_indices.push_back(target);
                if (graph->isWeighted()) {
                    const Edge* edge = graph->getEdge(edge_id);
                    weights.push_back(edge ? edge->getWeight() : 1.0f);
                }
                current_offset++;
            }
        }
    }
    row_offsets.push_back(current_offset);

    // Allocate GPU memory
    const size_t vertex_size = (graph->getVertexCount() + 1) * sizeof(vertex_id_t);
    const size_t edge_size = graph->getEdgeCount() * sizeof(vertex_id_t);
    const size_t weight_size = graph->isWeighted() ? graph->getEdgeCount() * sizeof(weight_t) : 0;

    // Free previous allocations
    if (gpu_data_->row_offsets) hipFree(gpu_data_->row_offsets);
    if (gpu_data_->column_indices) hipFree(gpu_data_->column_indices);
    if (gpu_data_->weights) hipFree(gpu_data_->weights);

    // Allocate new memory
    checkCudaError(hipMalloc(&gpu_data_->row_offsets, vertex_size),
                   "Failed to allocate GPU memory for row offsets");
    checkCudaError(hipMalloc(&gpu_data_->column_indices, edge_size),
                   "Failed to allocate GPU memory for column indices");
    
    if (graph->isWeighted()) {
        checkCudaError(hipMalloc(&gpu_data_->weights, weight_size),
                      "Failed to allocate GPU memory for weights");
    }

    // Copy data to GPU
    checkCudaError(hipMemcpy(gpu_data_->row_offsets, row_offsets.data(),
                             vertex_size, hipMemcpyHostToDevice),
                   "Failed to copy row offsets to GPU");
    checkCudaError(hipMemcpy(gpu_data_->column_indices, column_indices.data(),
                             edge_size, hipMemcpyHostToDevice),
                   "Failed to copy column indices to GPU");
    
    if (graph->isWeighted()) {
        checkCudaError(hipMemcpy(gpu_data_->weights, weights.data(),
                                 weight_size, hipMemcpyHostToDevice),
                      "Failed to copy weights to GPU");
    }

    // Store graph properties
    gpu_data_->num_vertices = graph->getVertexCount();
    gpu_data_->num_edges = graph->getEdgeCount();
    gpu_data_->is_directed = graph->isDirected();
    gpu_data_->is_weighted = graph->isWeighted();

    LOG_INFO("Graph transferred to GPU: {} vertices, {} edges",
             gpu_data_->num_vertices, gpu_data_->num_edges);
}

template<typename T>
void GPUEngine::parallelVertexOp(std::function<void(vertex_id_t, T&)> kernel,
                                std::vector<T>& vertex_data) {
    if (!initialized_ || !gpu_data_) {
        throw std::runtime_error("GPU engine not properly initialized");
    }

    // Allocate device memory for vertex data
    T* d_vertex_data;
    const size_t vertex_data_size = vertex_data.size() * sizeof(T);
    
    checkCudaError(hipMalloc(&d_vertex_data, vertex_data_size),
                   "Failed to allocate GPU memory for vertex data");

    // Copy input data to device
    checkCudaError(hipMemcpy(d_vertex_data, vertex_data.data(),
                             vertex_data_size, hipMemcpyHostToDevice),
                   "Failed to copy vertex data to GPU");

    // Calculate grid dimensions
    const int num_blocks = (gpu_data_->num_vertices + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    kernels::vertexKernel<<<num_blocks, THREADS_PER_BLOCK>>>(
        gpu_data_->row_offsets,
        gpu_data_->column_indices,
        gpu_data_->weights,
        d_vertex_data,
        gpu_data_->num_vertices
    );

    // Check for kernel launch errors
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    // Copy results back to host
    checkCudaError(hipMemcpy(vertex_data.data(), d_vertex_data,
                             vertex_data_size, hipMemcpyDeviceToHost),
                   "Failed to copy results from GPU");

    // Cleanup
    hipFree(d_vertex_data);
}

template<typename T>
void GPUEngine::parallelEdgeOp(std::function<void(edge_id_t, T&)> kernel,
                              std::vector<T>& edge_data) {
    if (!initialized_ || !gpu_data_) {
        throw std::runtime_error("GPU engine not properly initialized");
    }

    // Allocate device memory for edge data
    T* d_edge_data;
    const size_t edge_data_size = edge_data.size() * sizeof(T);
    
    checkCudaError(hipMalloc(&d_edge_data, edge_data_size),
                   "Failed to allocate GPU memory for edge data");

    // Copy input data to device
    checkCudaError(hipMemcpy(d_edge_data, edge_data.data(),
                             edge_data_size, hipMemcpyHostToDevice),
                   "Failed to copy edge data to GPU");

    // Calculate grid dimensions
    const int num_blocks = (gpu_data_->num_edges + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    kernels::edgeKernel<<<num_blocks, THREADS_PER_BLOCK>>>(
        gpu_data_->column_indices,
        gpu_data_->weights,
        d_edge_data,
        gpu_data_->num_edges
    );

    // Check for kernel launch errors
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    // Copy results back to host
    checkCudaError(hipMemcpy(edge_data.data(), d_edge_data,
                             edge_data_size, hipMemcpyDeviceToHost),
                   "Failed to copy results from GPU");

    // Cleanup
    hipFree(d_edge_data);
}

template<typename T>
T* GPUEngine::allocateDevice(size_t count) {
    T* device_ptr;
    checkCudaError(hipMalloc(&device_ptr, count * sizeof(T)),
                   "Failed to allocate GPU memory");
    return device_ptr;
}

template<typename T>
void GPUEngine::freeDevice(T* device_ptr) {
    if (device_ptr) {
        hipFree(device_ptr);
    }
}

template<typename T>
void GPUEngine::copyToDevice(const std::vector<T>& host_data, T* device_data) {
    checkCudaError(hipMemcpy(device_data, host_data.data(),
                             host_data.size() * sizeof(T), hipMemcpyHostToDevice),
                   "Failed to copy data to GPU");
}

template<typename T>
void GPUEngine::copyToHost(const T* device_data, std::vector<T>& host_data) {
    checkCudaError(hipMemcpy(host_data.data(), device_data,
                             host_data.size() * sizeof(T), hipMemcpyDeviceToHost),
                   "Failed to copy data from GPU");
}

bool GPUEngine::isGPUAvailable() const {
    int device_count;
    hipGetDeviceCount(&device_count);
    return device_count > 0;
}

int GPUEngine::getComputeCapability() const {
    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, device_id_),
                   "Failed to get device properties");
    return prop.major * 10 + prop.minor;
}

size_t GPUEngine::getAvailableMemory() const {
    size_t free_memory, total_memory;
    checkCudaError(hipMemGetInfo(&free_memory, &total_memory),
                   "Failed to get memory info");
    return free_memory;
}

void GPUEngine::checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::string error_message = std::string(message) + ": " + 
                                  hipGetErrorString(error);
        LOG_ERROR(error_message);
        throw std::runtime_error(error_message);
    }
}

bool GPUEngine::checkGPUCapabilities() {
    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, device_id_),
                   "Failed to get device properties");
    
    int compute_capability = prop.major * 10 + prop.minor;
    
    if (compute_capability < MIN_GPU_COMPUTE_CAPABILITY) {
        LOG_WARNING("GPU compute capability {}.{} is below minimum requirement {}.{}",
                   prop.major, prop.minor,
                   MIN_GPU_COMPUTE_CAPABILITY / 10,
                   MIN_GPU_COMPUTE_CAPABILITY % 10);
        return false;
    }
    
    return true;
}

// Explicit template instantiations for common types
template void GPUEngine::parallelVertexOp<float>(
    std::function<void(vertex_id_t, float&)>, std::vector<float>&);
template void GPUEngine::parallelVertexOp<double>(
    std::function<void(vertex_id_t, double&)>, std::vector<double>&);
template void GPUEngine::parallelVertexOp<int>(
    std::function<void(vertex_id_t, int&)>, std::vector<int>&);

template void GPUEngine::parallelEdgeOp<float>(
    std::function<void(edge_id_t, float&)>, std::vector<float>&);
template void GPUEngine::parallelEdgeOp<double>(
    std::function<void(edge_id_t, double&)>, std::vector<double>&);
template void GPUEngine::parallelEdgeOp<int>(
    std::function<void(edge_id_t, int&)>, std::vector<int>&);

template float* GPUEngine::allocateDevice<float>(size_t);
template double* GPUEngine::allocateDevice<double>(size_t);
template int* GPUEngine::allocateDevice<int>(size_t);

template void GPUEngine::freeDevice<float>(float*);
template void GPUEngine::freeDevice<double>(double*);
template void GPUEngine::freeDevice<int>(int*);

template void GPUEngine::copyToDevice<float>(const std::vector<float>&, float*);
template void GPUEngine::copyToDevice<double>(const std::vector<double>&, double*);
template void GPUEngine::copyToDevice<int>(const std::vector<int>&, int*);

template void GPUEngine::copyToHost<float>(const float*, std::vector<float>&);
template void GPUEngine::copyToHost<double>(const double*, std::vector<double>&);
template void GPUEngine::copyToHost<int>(const int*, std::vector<int>&);

// Add kernel template instantiations
namespace kernels {
    template __global__ void vertexKernel<float>(
        vertex_id_t*, vertex_id_t*, weight_t*, float*, size_t);
    template __global__ void vertexKernel<double>(
        vertex_id_t*, vertex_id_t*, weight_t*, double*, size_t);
    template __global__ void vertexKernel<int>(
        vertex_id_t*, vertex_id_t*, weight_t*, int*, size_t);
        
    template __global__ void edgeKernel<float>(
        vertex_id_t*, weight_t*, float*, size_t);
    template __global__ void edgeKernel<double>(
        vertex_id_t*, weight_t*, double*, size_t);
    template __global__ void edgeKernel<int>(
        vertex_id_t*, weight_t*, int*, size_t);
}

} // namespace graph_engine